#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_TEXT_SIZE 2000000
#define MAX_PATTERN_SIZE 100
#define MAX_PATTERNS 100
#define MAX_DISPLAY 20

typedef struct {
    char pattern[MAX_PATTERN_SIZE];
    int length;
    int hash;
} PatternInfo;

__device__ int compute_hash_gpu(char *str, int length, int d, int q) {
    int hash_value = 0;
    for (int i = 0; i < length; ++i) {
        hash_value = (d * hash_value + str[i]) % q;
    }
    return hash_value;
}

__global__ void rk_kernel(char *text, char *patterns, int *pattern_lengths, int *pattern_hashes, 
                          int *match_positions, int *match_pattern_ids, int *match_count,
                          int *pattern_match_counts, int text_length, int pattern_count, int d, int q) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= text_length) return;

    // Each thread handles a position in the text
    for (int p = 0; p < pattern_count; p++) {
        int pattern_length = pattern_lengths[p];
        int pattern_hash = pattern_hashes[p];
        
        // Skip if we don't have enough characters left
        if (i > text_length - pattern_length) continue;

        // Compute the hash of the current window
        int hash_value = 0;
        for (int j = 0; j < pattern_length; j++) {
            hash_value = (d * hash_value + text[i + j]) % q;
        }

        if (hash_value == pattern_hash) {
            bool match = true;
            for (int j = 0; j < pattern_length; j++) {
                if (text[i + j] != patterns[p * MAX_PATTERN_SIZE + j]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                // Increment total match count
                int idx = atomicAdd(match_count, 1);
                
                // Increment per-pattern match count
                atomicAdd(&pattern_match_counts[p], 1);
                
                // Store position and pattern ID for display
                if (idx < MAX_DISPLAY) {
                    match_positions[idx] = i;
                    match_pattern_ids[idx] = p;
                }
            }
        }
    }
}

char* read_text_file(const char* filename, int* length) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        printf("Error opening file %s\n", filename);
        exit(1);
    }

    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    rewind(file);

    char* buffer = (char*)malloc(size + 1);
    if (!buffer) {
        printf("Memory allocation failed\n");
        fclose(file);
        exit(1);
    }
    
    fread(buffer, 1, size, file);
    buffer[size] = '\0';
    fclose(file);

    *length = size;
    return buffer;
}

int read_patterns_file(const char* filename, PatternInfo* patterns) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        printf("Error opening patterns file %s\n", filename);
        exit(1);
    }

    char line[1024];
    if (fgets(line, sizeof(line), file) == NULL) {
        printf("Error reading patterns file or file is empty\n");
        fclose(file);
        exit(1);
    }
    fclose(file);

    // Remove newline if present
    int len = strlen(line);
    if (len > 0 && (line[len-1] == '\n' || line[len-1] == '\r')) {
        line[len-1] = '\0';
        len--;
    }
    if (len > 0 && line[len-1] == '\r') {
        line[len-1] = '\0';
    }

    // Parse the comma-separated patterns
    int pattern_count = 0;
    char* token = strtok(line, ",");
    while (token != NULL && pattern_count < MAX_PATTERNS) {
        strcpy(patterns[pattern_count].pattern, token);
        patterns[pattern_count].length = strlen(token);
        token = strtok(NULL, ",");
        pattern_count++;
    }

    return pattern_count;
}

int compute_hash_cpu(char *str, int length, int d, int q) {
    int hash_value = 0;
    for (int i = 0; i < length; ++i) {
        hash_value = (d * hash_value + str[i]) % q;
    }
    return hash_value;
}

int main() {
    const char* text_filename = "human_10m_upper.txt";
    const char* pattern_filename = "pattern.txt";
    int d = 256, q = 101;

    // Read text file
    int text_length;
    char* text = read_text_file(text_filename, &text_length);

    // Read patterns
    PatternInfo patterns[MAX_PATTERNS];
    int pattern_count = read_patterns_file(pattern_filename, patterns);
    
    printf("Loaded %d patterns from %s\n", pattern_count, pattern_filename);
    for (int i = 0; i < pattern_count; i++) {
        // Precompute hash for each pattern
        patterns[i].hash = compute_hash_cpu(patterns[i].pattern, patterns[i].length, d, q);
        printf("Pattern %d: %s (length: %d, hash: %d)\n", 
               i, patterns[i].pattern, patterns[i].length, patterns[i].hash);
    }

    // Prepare data for GPU
    char *d_text, *d_patterns;
    int *d_pattern_lengths, *d_pattern_hashes;
    int *d_match_positions, *d_match_pattern_ids, *d_match_count;
    int *d_pattern_match_counts;
    
    // Host memory for results
    int *match_positions = (int*)malloc(MAX_DISPLAY * sizeof(int));
    int *match_pattern_ids = (int*)malloc(MAX_DISPLAY * sizeof(int));
    int match_count = 0;
    int *pattern_match_counts = (int*)calloc(pattern_count, sizeof(int));

    // Prepare pattern data
    char* patterns_flat = (char*)malloc(pattern_count * MAX_PATTERN_SIZE);
    int* pattern_lengths = (int*)malloc(pattern_count * sizeof(int));
    int* pattern_hashes = (int*)malloc(pattern_count * sizeof(int));
    
    for (int i = 0; i < pattern_count; i++) {
        strcpy(&patterns_flat[i * MAX_PATTERN_SIZE], patterns[i].pattern);
        pattern_lengths[i] = patterns[i].length;
        pattern_hashes[i] = patterns[i].hash;
    }

    // Allocate device memory
    hipMalloc((void**)&d_text, text_length);
    hipMalloc((void**)&d_patterns, pattern_count * MAX_PATTERN_SIZE);
    hipMalloc((void**)&d_pattern_lengths, pattern_count * sizeof(int));
    hipMalloc((void**)&d_pattern_hashes, pattern_count * sizeof(int));
    hipMalloc((void**)&d_match_positions, MAX_DISPLAY * sizeof(int));
    hipMalloc((void**)&d_match_pattern_ids, MAX_DISPLAY * sizeof(int));
    hipMalloc((void**)&d_match_count, sizeof(int));
    hipMalloc((void**)&d_pattern_match_counts, pattern_count * sizeof(int));

    // Copy data to device
    hipMemcpy(d_text, text, text_length, hipMemcpyHostToDevice);
    hipMemcpy(d_patterns, patterns_flat, pattern_count * MAX_PATTERN_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_pattern_lengths, pattern_lengths, pattern_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern_hashes, pattern_hashes, pattern_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_match_count, &match_count, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_pattern_match_counts, 0, pattern_count * sizeof(int));

    // Timer to measure execution time
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (text_length + blockSize - 1) / blockSize;
    printf("Launching kernel with grid size: %d, block size: %d\n", gridSize, blockSize);
    
    rk_kernel<<<gridSize, blockSize>>>(d_text, d_patterns, d_pattern_lengths, d_pattern_hashes,
                                   d_match_positions, d_match_pattern_ids, d_match_count,
                                   d_pattern_match_counts, text_length, pattern_count, d, q);

    // Synchronize and measure time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy results from device to host
    hipMemcpy(match_positions, d_match_positions, MAX_DISPLAY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(match_pattern_ids, d_match_pattern_ids, MAX_DISPLAY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&match_count, d_match_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pattern_match_counts, d_pattern_match_counts, pattern_count * sizeof(int), hipMemcpyDeviceToHost);

    // Check for CUDA errors
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipError_t));
    }

    // Output results
    printf("\nSample matches found:\n");
    for (int i = 0; i < match_count && i < MAX_DISPLAY; i++) {
        printf("Pattern '%s' found at index %d\n", 
               patterns[match_pattern_ids[i]].pattern, match_positions[i]);
    }
    
    if (match_count > MAX_DISPLAY) {
        printf("... (showing only first %d matches out of %d)\n", MAX_DISPLAY, match_count);
    }
    
    // Print per-pattern match counts
    printf("\nMatch counts per pattern:\n");
    int total_matches = 0;
    for (int i = 0; i < pattern_count; i++) {
        printf("Pattern '%s': %d matches\n", patterns[i].pattern, pattern_match_counts[i]);
        total_matches += pattern_match_counts[i];
    }
    
    printf("Total matches found across all patterns: %d\n", total_matches);
    printf("Execution time: %.2f ms\n", milliseconds);

    // Free memory
    free(text);
    free(patterns_flat);
    free(pattern_lengths);
    free(pattern_hashes);
    free(match_positions);
    free(match_pattern_ids);
    free(pattern_match_counts);
    
    hipFree(d_text);
    hipFree(d_patterns);
    hipFree(d_pattern_lengths);
    hipFree(d_pattern_hashes);
    hipFree(d_match_positions);
    hipFree(d_match_pattern_ids);
    hipFree(d_match_count);
    hipFree(d_pattern_match_counts);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}